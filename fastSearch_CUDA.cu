#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "minHeap.c"
#include <assert.h>


#define CUDACHECK(cmd) do { \
    hipError_t e = cmd;    \
    if( e != hipSuccess ) { \
    printf("Failed: Cuda error %s:%d '%s'\n", \
        __FILE__,__LINE__,hipGetErrorString(e)); \
    exit(EXIT_FAILURE);     \
  } \
} while(0)

void swap(float *lhs, float *rhs)
{
    if (lhs == rhs)
        return;

    float tmp = *lhs;
    *lhs = *rhs;
    *rhs = tmp;
}

__global__ void fastJacc(unsigned int *allMols, unsigned int *queries, float *sims, int row, int size)
{
    // take steps over each fingerprint in allmols
    // aka grid-stride loop
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // the cardinality is stored as the first int, always.
    int cardX = queries[65 * row];
    for (int i = index * 65; i < size * 65; i += 65 * stride) {
        int totalSize = 0;
        // unroll the loop into 16 loops of 4. 15% speedup.
#pragma unroll 4
        for (int j = 1; j < 65; j++)
        {
            unsigned int x = queries[(65 * row) + j];
            unsigned int y = allMols[i + j];
            totalSize += __popc(x & y);
        }
        int cardY = allMols[i];
        // alternate eq for jaccard score: (int(x, y) / |x| + |y| + int(x, y)).
        int jaccDenom = cardX + cardY - totalSize;
        // theoretically can have div by 0 error, however, jaccard score isn't defined for int(x, y) = 0,
        // and the chances of it happening are basically 0.
        // so don't waste the compute power to guard against it. if you're getting errors here,
        // your data is probably corrupted.
        sims[(size * row) + (i / 65)] = float(totalSize) / jaccDenom;
    }
}

// function to get the topk from an array of similarities
void populateMinHeap(float *array, minHeap *m, int size) {
    for (int i = 0; i < size; i++) {
        if (m->size < m->maxSize) {
            insertNode(m, array[i], i);
        }

        else if (array[i] > getMinNode(m)) {
            // heapify instead?
            deleteNode(m);
            insertNode(m, array[i], i);
        }
    }
}

int main(int argc, char* argv[])
{
    if (argc != 6) {
        printf("%s\n", "Must input 4 arguements, in order, input database, input queries, output file, int block size, int top_k.");
        printf("%s\n", "Example run: fastSearch_CUDA mols.bin inq.bin search.txt 1024 30");
        exit(EXIT_FAILURE);
    }

    FILE *database_ptr;
    database_ptr = fopen(argv[1], "rb"); // r for read, b for binary
    if (database_ptr == NULL) {
        printf("Couldn't find input database. Is the path correct?");
        exit(EXIT_FAILURE);
    }
    FILE *queries_ptr;
    queries_ptr = fopen(argv[2], "rb"); // r for read, b for binary
    if (queries_ptr == NULL) {
        printf("Couldn't find input queries. Is the path correct?");
        exit(EXIT_FAILURE);
    }

    // start reading the database file
    int size;
    // read num mols
    fread(&size, sizeof(int), 1, database_ptr);
    unsigned int *mols;
    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&mols, sizeof(int) * size * (64 + 1));
    // actually read the file to unified memory
    int freadReturnVal = fread(mols, sizeof(int) * (64 + 1), size, database_ptr);
    if (freadReturnVal != size) {
        printf("%s", "Bad read on input database. Do you have enough memory? Is the file corrupted? Has it been preprocessed with convert_to_binary?");
        exit(EXIT_FAILURE);
    }
    fclose(database_ptr);

    // exact same as above
    int num_queries;
    fread(&num_queries, sizeof(int), 1, queries_ptr); // read num queries
    unsigned int *queries;
    hipMallocManaged(&queries, sizeof(int) * num_queries * (64 + 1));
    freadReturnVal = fread(queries, sizeof(int) * (64 + 1), num_queries, queries_ptr);
    if (freadReturnVal != num_queries) {
        printf("%s", "Bad read on input queries. Do you have enough memory? Is the file corrupted? Has it been preprocessed with convert_to_binary?");
        exit(EXIT_FAILURE);
    }
    fclose(queries_ptr);

    int numToRun = num_queries;

    printf("%s", "Number of queries: ");
    printf("%d\n", num_queries);
    int k = atoi(argv[5]);

    clock_t start, end;
    double cpu_time_used;
    start = clock();
    float *sims;
    // width * height -- CUDA doesn't like 2d arrays
    hipMallocManaged(&sims, (size * sizeof(float)) * (numToRun));
    // this value should probably be about 1024
    int blockSize = atoi(argv[4]);
    int numBlocks = (size + blockSize - 1) / blockSize;
    for (int i = 0; i < numToRun; i++)
    {
        fastJacc << <numBlocks, blockSize >> > (mols, queries, sims, i, size);
    }
    CUDACHECK(hipDeviceSynchronize());

    struct minHeap **heaps = (struct minHeap**) malloc(sizeof(struct minHeap *) * numToRun);
    for (int i = 0; i < numToRun; i++) {
        heaps[i] = initMinHeap(k);
        populateMinHeap(&sims[(i * size)], heaps[i], size);
        // uncomment these if you want to look at your data
        // preorderTraversal(heaps[i], 0);
        // printf("%s\n", "");
    }

    end = clock();
    cpu_time_used = (((double)(end - start)) / CLOCKS_PER_SEC);
    printf("%s", "Time to run queries: ");
    printf("%f\n", cpu_time_used);

    FILE *out = fopen(argv[3], "w");
    for (int i = 0; i < numToRun; i++) {
        minHeap *currHeap = heaps[i];
        for (int j = 0; j < k; j++) {
            fprintf(out, "%s", "(");
            fprintf(out, "%.4f", currHeap->elem[j].data);
            fprintf(out, "%s", ", ");
            fprintf(out, "%d", currHeap->elem[j].idx);
            fprintf(out, "%s", ")");
            if (j + 1 < size)
                fprintf(out, "%s", ", ");
        }
        fprintf(out, "%s", "\n");
    }

    fclose(out);

    // Free memory
    free(heaps);
    hipFree(queries);
    hipFree(mols);
    hipFree(sims);
    return 0;
}